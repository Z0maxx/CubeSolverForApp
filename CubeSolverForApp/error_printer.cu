#include "error_printer.cpp"

void printError(hipError_t error)
{
	printf("%s %s\n", hipGetErrorName(error), hipGetErrorString(error));
}